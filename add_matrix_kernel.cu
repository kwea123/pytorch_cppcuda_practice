#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


template <typename scalar_t>
__device__ __forceinline__ scalar_t identity(scalar_t z) {
    return z;
}


template <typename scalar_t>
__global__ void add_matrix_kernel_forward(
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> A,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> B,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> out
){    
    const int n = blockIdx.y;
    const int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (c < A.size(1)){
        out[n][c] = identity(A[n][c] + B[n][c]);
    }
}


torch::Tensor add_matrix_cu_forward(
  torch::Tensor A,
  torch::Tensor B
){
    torch::Tensor out = torch::zeros_like(A);

    const int n_row = A.size(0);
    const int n_col = A.size(1);
    const int threads = 1024;
    const dim3 blocks((n_col + threads - 1) / threads, n_row); // to cover all elements
    
    // instantiate kernel
    AT_DISPATCH_FLOATING_TYPES(A.type(), "add_matrix_cu_forward", 
    ([&] {
        add_matrix_kernel_forward<scalar_t><<<blocks, threads>>>(
            A.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            B.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            out.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    })
    );
    return out;
}